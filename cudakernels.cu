#include "hip/hip_runtime.h"
#include "cudakernels.h"
#include <stdio.h>
#include "vel.h"
#include <time.h>

#ifndef __params__
#define Z_MAX 10
#define DIM 2
#endif

#ifndef __constants__
#define kB 1.38064852e-5					// Boltzmann constant
#define b 0.1								// Persistence length
#define T 300 								// Temperature
#define ae 0.1 								// Strength of bond - includes activation energy
#define delxe 0.15 							// parameter for breaking crosslink connection
#define TIME_STEP 1e-4
#endif

__device__ static float vel[2] = {vel_x, vel_y};


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ float kfe_cuda(float force){
	return ae*expf(force*delxe/kB/T)*TIME_STEP;
}

__device__ float force_wlc_cuda(float x, float L){
	float t = x / L;
	if (t < 0.99){ return kB*T / b * (t + 1.0 / 4.0 / powf((1.0 - t), 2.0) - 1.0 / 4.0); }
	else { return 999999.0; }
}

__global__ void optimize_cuda(float* R, int* edges, float* damage_integral, float* forces, \
	const float* chain_len, const int num_nodes, const int num_edges, \
	const bool* PBC_STATUS, const float* PBC_vector, \
	const int* tnodes, int n_tnodes, const int* moving_nodes, int n_moving, \
	float* plate_force, int n_steps,\
	float eta = 0.01, float alpha = 0.9, int max_iter_opt = 1000){

	int tid = threadIdx.x + blockIdx.x*BLOCK_SIZE;
	float pbc_v[2] = {PBC_vector[0], PBC_vector[1]}; 
	float rms_history[2];
	float delR[2];
	float grad[2];

	int pair, n1, n2, n_t;
	float L, x1, x2, y1, y2, dist, force;
	float unitvector[DIM];
	bool pbc;

	//if(tid==1){printf}

	if(tid<num_edges){
		damage_integral[tid] = 0.0;
		pbc = PBC_STATUS[tid];

		// Assign threads to edges
		pair = tid * 2;
		L = chain_len[tid];

		// read the nodes that the thread has been assigned
		n1 = edges[pair];
		n2 = edges[pair+1];
	}
	__syncthreads();

	for(int iter = 0; iter<n_steps; iter++){
		
		// reset grads for next iteration
		rms_history[0] = 0.0;
		rms_history[1] = 0.0;
		delR[0] = 0.0;
		delR[1] = 0.0;
		grad[0] = 0.0;
		grad[1] = 0.0;

		for(int step = 0; step < max_iter_opt; step++){
			///////////////////////////////////////////////
			// Force calculations
			// 
			// Here each edge is assigned one thread
			//
			///////////////////////////////////////////////
	
			// zero all forces
			if(tid<num_nodes*DIM){
				forces[tid] = 0.0;
			}
			__syncthreads();

			force = 0.0;
					
			if(tid < num_edges){
			// Check if connection exists 
			if(n1 != SPCL_NUM && n2 != SPCL_NUM){

				// read the positions of the crosslinkers
				x1 = R[n1*DIM];
				y1 = R[n1*DIM + 1];
				x2 = R[n2*DIM];
				y2 = R[n2*DIM + 1];
				//if(step==0 && iter==0){printf("(%f, %f); (%f, %f)\n", x1, y1, x2, y2);}
				
				// Calculate distance, unit vector and force
				// Shared memory is per block. If num_edges*DIM is too large each block
				// can be held responsible for separate pairs and then atomic adds can 
				// be done. Another approach could be to have each thread implement force
				// calc for one node to avoid atomic adds but that requires each thread to 
				// run through all edges and figure out which ones to add. That will be order
				// n whereas atomic adds should be order z extra work

				// check for PBC;
				if(pbc==true){
					dist = hypotf(x1-x2-pbc_v[0], y1-y2-pbc_v[1]);
					// add unitvector 3 for DIM 3. __in future use for loop here
					unitvector[0] = (x1 - x2 - pbc_v[0])/dist;
					unitvector[1] = (y1 - y2 - pbc_v[1])/dist;
				}
				else{
					dist = hypotf(x1-x2, y1-y2);
					// add unitvector 3 for DIM 3. __in future use for loop here
					unitvector[0] = (x1 - x2)/dist;
					unitvector[1] = (y1 - y2)/dist;
				}
			
				
				// calculate force
				force = force_wlc_cuda(dist, L);

				// Break if force too high
				if(force==999999){
					//printf("Breaking bond between %d and %d at iter %d, step %d\n", n1, n2, iter, step);
					n1 = SPCL_NUM;
					n2 = SPCL_NUM;
					edges[pair] = n1;
					edges[pair + 1] = n2;
					force = 0.0;
					damage_integral[tid] = 1.1;
				}
				else{
					// add the forces calculated to the nodes (atomic add)
					atomicAdd(&forces[n1*DIM], -1.0*force*unitvector[0]);
					atomicAdd(&forces[n1*DIM+1], -1.0*force*unitvector[1]);

					atomicAdd(&forces[n2*DIM], force*unitvector[0]);
					atomicAdd(&forces[n2*DIM+1], force*unitvector[1]);
				}
				
			}}
			__syncthreads();
			
			/////////////////////////////////////////////////
			//
			// Optimization step
			//
			/////////////////////////////////////////////////

			// Assign each thread to nodes*DIM
			if(tid<n_moving){
				n_t = moving_nodes[tid];
				grad[0] = forces[n_t*DIM];
				grad[1] = forces[n_t*DIM + 1];

				rms_history[0] = alpha*rms_history[0] + (1-alpha)*grad[0]*grad[0];
				rms_history[1] = alpha*rms_history[1] + (1-alpha)*grad[1]*grad[1];

				delR[0] = eta/__frsqrt_rn((rms_history[0] + 1.0e-6)) * grad[0];
				delR[1] = eta/__frsqrt_rn((rms_history[1] + 1.0e-6)) * grad[1];
				
				R[n_t*DIM] += delR[0];
				//if(fabs(delR[0])>10.0){printf("For node %d we have forces %0.3f\n",n_t, grad[0] );}
				R[n_t*DIM + 1] += delR[1];
				//if(fabs(delR[1])>10.0){printf("For node %d we have forces %0.3f\n",n_t, grad[1] );}
			}
			__syncthreads();
		}


		// Check if connection exists 
		if(tid<num_edges){
			if(damage_integral[tid]<1.0){
				// Update damage integral
				damage_integral[tid] += kfe_cuda(force)*TIME_STEP;
			}
			// Update edges acc. to damage
			if(damage_integral[tid] > 1.0){
				damage_integral[tid] = 1.1;
				n1 = SPCL_NUM;
				n2 = SPCL_NUM;
				edges[pair] = SPCL_NUM;
				edges[pair + 1] = SPCL_NUM;
			}
		}

		// update the force in the array
		if(tid<n_tnodes){
			n_t = tnodes[tid];
			atomicAdd(&plate_force[iter*DIM], forces[DIM*n_t]);
			atomicAdd(&plate_force[iter*DIM + 1], forces[DIM*n_t + 1]);
		}

		// move top nodes acc. to velocity
		if(tid < 2*n_tnodes && tid >= n_tnodes){
			n_t = tnodes[tid - n_tnodes];
			R[DIM*n_t] += vel[0]*TIME_STEP;
			R[DIM*n_t + 1] += vel[1]*TIME_STEP;
		}
		__syncthreads();
	
	}
}


void sanity_check(hostvars* vars){
	int n1, n2,c=0;
	float dist, x1, x2, y1, y2, L;
	float pbc_v[2] = {vars->PBC_vector[0], vars->PBC_vector[1]};
	for(int i=0; i<vars->n_elems; i++){
		n1 = vars->edges[2*i];
		n2 = vars->edges[2*i+1];
		x1 = vars->R[n1*DIM];
		y1 = vars->R[n1*DIM + 1];
		x2 = vars->R[n2*DIM];
		y2 = vars->R[n2*DIM + 1];
		if(vars->PBC[i] == true){x2 += pbc_v[0]; y2 += pbc_v[1]; }
		dist = sqrtf((x1-x2)*(x1-x2) + (y1-y2)*(y1-y2));
		L = vars->L[i];
		if (dist>=L){
			printf("(%0.2f, %0.2f); (%0.2f, %0.2f) and L = %0.2f\n", x1, y1, x2, y2, L);
			c += 1;
		}
		}
	printf("%d of %d elements make no sense!", c, vars->n_elems);
}

void pull_CUDA(hostvars* vars, int n_steps){
	// Pass all host variables in a struct
	
	// Initialize nodes and edges
	float* R_d; int* edges_d;
	float* forces_d;
	int* tsideNodes_d;
	int* moving_nodes_d;
	bool* PBC_d;
	float* L_d; float* damage_d;
	float* pull_forces_d;
	float* PBC_vector_d;
	int n_nodes = vars->n_nodes;
	int n_elems = vars->n_elems;
	int n_tside = vars->n_tnodes;
	int n_moving = vars->n_moving;
	
	//sanity_check(vars);

	// Check if the transfers are ok
	printf("n_nodes: %d \t", n_nodes);
	printf("n_elems: %d \t", n_elems);
	printf("n_tnodes: %d \t", n_tside);
	printf("n_moving: %d \n", n_moving);

	// GPU allocations
	hipMalloc((void**)&R_d, n_nodes*DIM*sizeof(float));
	hipMalloc((void**)&forces_d, n_nodes*DIM*sizeof(float));
	hipMalloc((void**)&edges_d, 2*n_elems*sizeof(int));
	hipMalloc((void**)&tsideNodes_d, n_tside*sizeof(int));
	hipMalloc((void**)&moving_nodes_d, n_moving*sizeof(int));
	hipMalloc((void**)&PBC_d, n_elems*sizeof(bool));
	hipMalloc((void**)&PBC_vector_d, DIM*sizeof(float));
	hipMalloc((void**)&L_d, n_elems*sizeof(float));
	hipMalloc((void**)&damage_d, n_elems*sizeof(float));
	hipMalloc((void**)&pull_forces_d, n_steps*DIM*sizeof(float));
	printf("Malloc successful! \n");

	// Copy host to device
	hipMemcpy(PBC_vector_d, vars->PBC_vector, DIM*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(PBC_d, vars->PBC, n_elems*sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(R_d, vars->R, n_nodes*DIM*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(forces_d, vars->forces, n_nodes*DIM*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(edges_d, vars->edges, 2*n_elems*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(moving_nodes_d, vars->moving_nodes, n_moving*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(tsideNodes_d, vars->tsideNodes, n_tside*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(L_d, vars->L, n_elems*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(damage_d, vars->damage, n_elems*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pull_forces_d, vars->pull_forces, n_steps*DIM*sizeof(float), hipMemcpyHostToDevice);
	printf("Transfer successful! \n");

	// Define grid and block size
	// Launch atleast as many threads as edges
	dim3 gridsize((n_elems-1)/BLOCK_SIZE + 1);
	dim3 blocksize(BLOCK_SIZE);

	// Launch timer code
	clock_t t = clock();

	int n_e = 0;
	for(int i=0; i< n_elems; i++){
		if(vars->edges[2*i]!=SPCL_NUM){
			n_e++;
		}
	}
	printf("We have %d edges\n",n_e);

	printf("Launching kernel...\n");
	optimize_cuda<<< gridsize, blocksize >>>(
		R_d, edges_d, damage_d, forces_d, \
		L_d, n_nodes, n_elems, PBC_d, \
		PBC_vector_d, tsideNodes_d, n_tside, \
		moving_nodes_d, n_moving, \
		pull_forces_d, n_steps);
	hipDeviceSynchronize();
	//gpuErrchk(hipDeviceSynchronize());
	printf("%d steps took %0.5f s\n", n_steps, float(clock()-t)/CLOCKS_PER_SEC);

	// Copy device to host
	hipMemcpy(vars->R, R_d,  n_nodes*DIM*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(vars->forces, forces_d,  n_nodes*DIM*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(vars->damage, damage_d,  n_elems*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(vars->pull_forces, pull_forces_d,  n_steps*DIM*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(vars->edges, edges_d,  2*n_elems*sizeof(int), hipMemcpyDeviceToHost);

	n_e = 0;
	for(int i=0; i< n_elems; i++){
		if(vars->edges[2*i]!=SPCL_NUM || vars->edges[2*i+1]!=SPCL_NUM){
			n_e++;
		}
	}
	printf("We have %d edges\n",n_e);

	// for(int i=0; i<n_elems; i++){
	// 	if (vars->damage[i]>=1.0){
	// 		printf("damage[%d]\t%0.5f\n",i, vars->damage[i]);}
	// 	// for(int d=0; d<DIM; d++){
	// 	// 	printf("%0.3f\t",vars->R[i*DIM + d]);
	// 	// }
	// 	// printf("\n");
	// }

	// Free up global memory
	hipFree(R_d);
	hipFree(forces_d);
	hipFree(edges_d);
	hipFree(moving_nodes_d);
	hipFree(tsideNodes_d);
	hipFree(PBC_d);
	hipFree(PBC_vector_d);
	hipFree(L_d);
	hipFree(damage_d);
	hipFree(pull_forces_d);

	return;
}
