#include "hip/hip_runtime.h"
#include "cudakernels.h"
#include <stdio.h>
#include "vel.h"
#include <time.h>

#ifndef __params__
#define Z_MAX 10
#define DIM 2
#endif

#ifndef __constants__
#define kB 1.38064852e-5					// Boltzmann constant
#define b 0.1								// Persistence length
#define T 300 								// Temperature
#define ae 0.1 								// Strength of bond - includes activation energy
#define delxe 0.15 							// parameter for breaking crosslink connection
#define TIME_STEP 1e-4
#endif

__device__ static float vel[2] = {vel_x, vel_y};


__device__ float kfe_cuda(float force){
	return expf(force*delxe/kB/T)*TIME_STEP;
}

__device__ float force_wlc_cuda(float x, float L){
	float t = x / L;
	if (t < 0.99){ return kB*T / b * (t + 1.0 / 4.0 / powf((1.0 - t), 2.0) - 1.0 / 4.0); }
	else { return 999999.0; }
}

__device__ bool notmember(int id, const int* bnodes, int n_bnodes, \
	const int* tnodes, int n_tnodes){
	bool out = false;
	for(int i=0;i<max(n_tnodes, n_bnodes);i++){
		if(i<n_tnodes){
			if(id==tnodes[i]){out=true; break;}
		}
		if(i<n_bnodes){
			if(id==bnodes[i]){out=true; break;}
		}
	}
	return out;
}

__global__ void optimize_cuda(float*R, int* edges, float* damage_integral, float* forces, \
	const float* chain_len, const int num_nodes, const int num_edges, \
	const bool* PBC_STATUS, const float* PBC_vector, \
	const int* tnodes, int n_tnodes, const int* moving_nodes, int n_moving, \
	float* plate_force, int n_steps,\
	float eta = 0.1, float alpha = 0.9, int max_iter_opt = 1000){

	// Get indices
	int tx = threadIdx.x; 
	int bx = blockIdx.x;
	int tid = tx + bx*BLOCK_SIZE;
	

	float rms_history[2] = {0.0, 0.0};
	float delR[2] = {0.0, 0.0};
	float grad[2] = {0.0, 0.0};

	int pair, edge_num, n1, n2, n_t;
	float L, x1, x2, y1, y2, dist, force, diss_energy, top_force_x, top_force_y;
	float unitvector[DIM];
	for(int iter = 0; iter<n_steps; iter++){
		for(int step = 0; step < max_iter_opt; step++){
			///////////////////////////////////////////////
			// Force calculations
			// 
			// Here each edge is assigned one thread
			//
			///////////////////////////////////////////////

			// Assign threads to edges
			pair = tid * 2;
			edge_num = tid; 
			if(tid<num_edges){
			L = chain_len[edge_num];

			// read the nodes that the thread has been assigned
			n1 = edges[pair];
			n2 = edges[pair+1];

			// Check if connection exists 
			if(n1!=SPCL_NUM || n2 != SPCL_NUM){
				// read the positions of the crosslinkers
				x1 = R[n1*DIM];
				y1 = R[n1*DIM + 1];
				x2 = R[n2*DIM];
				y2 = R[n2*DIM + 1];

				// Calculate distance, unit vector and force
				// Shared memory is per block. If num_edges*DIM is too large each block
				// can be held responsible for separate pairs and then atomic adds can 
				// be done. Another approach could be to have each thread implement force
				// calc for one node to avoid atomic adds but that requires each thread to 
				// run through all edges and figure out which ones to add. That will be order
				// n whereas atomic adds should be order z extra work

				// check for PBC;
				if(PBC_STATUS[edge_num]==true){
					dist = hypot(x1-x2-PBC_vector[0], y1-y2-PBC_vector[1]);
				}
				else{
					dist = hypot(x1-x2, y1-y2);
				}

				// add unitvector 3 for DIM 3. __in future use for loop here
				unitvector[0] = (x1 - x2)/dist;
				unitvector[1] = (y1 - y2)/dist;


				// calculate force
				force = force_wlc_cuda(dist, L);

				// zero all forces
				if(tid<num_nodes*DIM){
					forces[tid] = 0.0;
				}

				//required before next step
				__syncthreads();

				// add the forces calculated to the nodes (atomic add)
				atomicAdd(&forces[n1], force*unitvector[0]);
				atomicAdd(&forces[n1+1], force*unitvector[1]);

				atomicAdd(&forces[n2], force*unitvector[0]);
				atomicAdd(&forces[n2+1], force*unitvector[1]);

				__syncthreads();
			}

			/////////////////////////////////////////////////
			//
			// Optimization step
			//
			/////////////////////////////////////////////////

			// Assign each thread to nodes*DIM
			if(tid<n_moving){
				n1 = moving_nodes[tid];
				grad[0] = forces[n1*2];
				grad[1] = forces[n1*2 + 1];
				
				rms_history[0] = alpha*rms_history[0] + (1-alpha)*grad[0]*grad[0];
				rms_history[1] = alpha*rms_history[1] + (1-alpha)*grad[1]*grad[1];

				delR[0] = eta*__frsqrt_rn(1.0/(rms_history[0] + 1.0e-6)) * grad[0];
				delR[1] = eta*__frsqrt_rn(1.0/(rms_history[1] + 1.0e-6)) * grad[1];
				
				R[n1*2] += delR[0];
				R[n1*2 + 1] += delR[1];
			}
			__syncthreads();
		}

		// Update damage integral
		diss_energy = kfe_cuda(force)*TIME_STEP;
		damage_integral[edge_num] += diss_energy;

		// Update edges acc. to damage
		if(damage_integral[edge_num] > 1.0){
			edges[pair] = SPCL_NUM;
			edges[pair + 1] = SPCL_NUM;
		}

		// update the force in the array
		if(tid<n_tnodes){
			n_t = tnodes[tid];
			top_force_x = forces[DIM*n_t];
			top_force_y = forces[DIM*n_t + 1];
			atomicAdd(&plate_force[iter*DIM], top_force_x);
			atomicAdd(&plate_force[iter*DIM + 1], top_force_y);
		}

		// move top nodes acc. to velocity
		if(tid < 2*n_tnodes && tid >= n_tnodes){
			n_t = tnodes[tid - n_tnodes];
			R[DIM*n_t] += vel[0]*TIME_STEP;
			R[DIM*n_t + 1] += vel[1]*TIME_STEP;
		}
		__syncthreads();
	}	
	//	if(iter%500 == 0 && tid == 1){
	//		printf("Completed %d iterations...\n",iter);
	//		printf("%d took %0.5f s\n", iter, float(clock()-t)/CLOCKS_PER_SEC);
	//		t = clock();
	//	}
	}
}


void pull_CUDA(hostvars* vars, int n_steps){
	// Pass all host variables in a struct
	
	// Initialize nodes and edges
	printf("Got in! \n");
	float* R_d; int* edges_d;
	float* forces_d;
	int* tsideNodes_d;
	int* moving_nodes_d;
	bool* PBC_d;
	float* L_d; float* damage_d;
	float* pull_forces_d;
	float* PBC_vector_d;
	int n_nodes = vars->n_nodes;
	int n_elems = vars->n_elems;
	int n_tside = vars->n_tnodes;
	int n_moving = vars->n_moving;
	
	// Check if the transfers are ok
	printf("n_nodes: %d \t", n_nodes);
	printf("n_elems: %d \t", n_elems);
	printf("n_tnodes: %d \t", n_tside);
	printf("n_moving: %d \n", n_moving);

	// GPU allocations
	hipMalloc((void**)&R_d, n_nodes*DIM*sizeof(float));
	hipMalloc((void**)&forces_d, n_nodes*DIM*sizeof(float));
	hipMalloc((void**)&edges_d, Z_MAX*n_nodes*2*sizeof(int));
	hipMalloc((void**)&tsideNodes_d, n_tside*sizeof(int));
	hipMalloc((void**)&moving_nodes_d, n_moving*sizeof(float));
	hipMalloc((void**)&PBC_d, 2*n_elems*sizeof(bool));
	hipMalloc((void**)&PBC_vector_d, DIM*sizeof(float));
	hipMalloc((void**)&L_d, n_elems*sizeof(float));
	hipMalloc((void**)&damage_d, n_elems*sizeof(float));
	hipMalloc((void**)&pull_forces_d, n_steps*DIM*sizeof(float));
	printf("Malloc successful! \n");

	// Copy host to device
	hipMemcpy(PBC_vector_d, vars->PBC_vector, DIM*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(PBC_d, vars->PBC, n_elems*sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(R_d, vars->R, n_nodes*DIM*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(forces_d, vars->forces, n_nodes*DIM*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(edges_d, vars->edges, 2*n_elems*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(moving_nodes_d, vars->moving_nodes, n_moving*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(tsideNodes_d, vars->tsideNodes, n_tside*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(L_d, vars->L, 2*n_elems*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(damage_d, vars->damage, 2*n_elems*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pull_forces_d, vars->pull_forces, n_steps*DIM*sizeof(float), hipMemcpyHostToDevice);
	printf("Transfer successful! \n");
	
	// Define grid and block size
	// Launch atleast as many threads as edges
	dim3 gridsize((n_elems-1)/BLOCK_SIZE + 1);
	dim3 blocksize(BLOCK_SIZE);

	// Launch timer code
	clock_t t = clock();

	printf("Launching kernel...\n");
	optimize_cuda<<< gridsize, blocksize >>>(
		R_d, edges_d, damage_d, forces_d, \
		L_d, n_nodes, n_elems, PBC_d, \
		PBC_vector_d, tsideNodes_d, n_tside, \
		moving_nodes_d, n_moving, \
		pull_forces_d, n_steps);
	hipDeviceSynchronize();
	
	printf("%d steps took %0.5f s\n", n_steps, float(clock()-t)/CLOCKS_PER_SEC);

	// Copy device to host
	hipMemcpy(vars->R, R_d,  n_nodes*DIM*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(vars->forces, forces_d,  n_nodes*DIM*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(vars->damage, damage_d,  2*n_elems*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(vars->pull_forces, pull_forces_d,  n_steps*DIM*sizeof(float), hipMemcpyDeviceToHost);

	// Free up global memory
	hipFree(R_d);
	hipFree(forces_d);
	hipFree(edges_d);
	hipFree(moving_nodes_d);
	hipFree(tsideNodes_d);
	hipFree(PBC_d);
	hipFree(PBC_vector_d);
	hipFree(L_d);
	hipFree(damage_d);
	hipFree(pull_forces_d);

	return;
}
